#include "hip/hip_runtime.h"
/**
 * @private
 * @headerfile peach.cuh <peach.cuh>
 * @copyright Adequate Systems LLC, 2018-2022. All Rights Reserved.
 * <br />For license information, please refer to ../LICENSE.md
 * @note A note on variable naming in this file...
 * [2] suffix implies dual stream variables; for maximum GPU utilisation
 * h_ prefix implies host memory; requires hipHostMalloc()
 * c_ prefix implies (constant) device memory
 * g_ prefix implies (global) device memory
 * d_ prefix implies device memory
 * *h_ pointers require hipHostMalloc();
 * *d_/*g_ pointers require hipMalloc();
*/

/* include guard */
#ifndef MOCHIMO_PEACH_CU
#define MOCHIMO_PEACH_CU


#include "extint.h"
#include "extmath.h"
#include "extprint.h"

#include "peach.cuh"

/* hashing functions used by Peach's nighthash */
#include "blake2b.cu"
#include "md2.cu"
#include "md5.cu"
#include "sha1.cu"
#include "sha256.cu"
#include "sha3.cu"

/**
 * @private
 * Definitions for embedding strings.
*/
#define cuSTRING(x) #x
#define cuSTR(x) cuSTRING(x)

/**
 * @private
 * Peach specific cuda error checking definition.
 * Destroys/frees resources on error.
*/
#define cuCHK(_cmd, _dev, _exec) \
   do { \
      hipError_t _cerr = _cmd; \
      if (_cerr != hipSuccess) { \
         int _n; hipGetDevice(&_n); \
         const char *_err = hipGetErrorString(_cerr); \
         pfatal("CUDA#%d->%s: %s", _n, cuSTR(_cmd), _err); \
         if (_dev != NULL) { \
            hipDeviceSynchronize(); \
            DEVICE_CTX *_d = _dev; \
            plog("CUDA#%d->state[%d:%d:%d] <<<%d, %d>>>: %d", _n, _d->id, \
               _d->type, _d->status, _d->grid, _d->block, _d->threads); \
            _d->status = DEV_FAIL; \
            PEACH_CUDA_CTX *_p = &PeachCudaCTX[_d->id]; \
            if (_p->stream[0]) hipStreamDestroy(_p->stream[0]); \
            if (_p->stream[1]) hipStreamDestroy(_p->stream[1]); \
            if (_p->h_solve) hipHostFree(_p->h_solve); \
            if (_p->h_ictx) hipHostFree(_p->h_ictx); \
            if (_p->d_solve[0]) hipFree(_p->d_solve[0]); \
            if (_p->d_solve[1]) hipFree(_p->d_solve[1]); \
            if (_p->d_ictx[0]) hipFree(_p->d_ictx[0]); \
            if (_p->d_ictx[1]) hipFree(_p->d_ictx[1]); \
            if (_p->d_map) hipFree(_p->d_map); \
         } \
         _exec; \
      } \
   } while(0)

/**
 * @private
 * Peach CUDA context. Managed internally by cross referencing parameters
 * of DEVICE_CTX passed to functions.
*/
typedef struct {
   nvmlDevice_t nvml_device;           /**< nvml device for monitoring */
   hipStream_t stream[2];             /**< asynchronous streams */
   SHA256_CTX *h_ictx[2], *d_ictx[2];  /**< sha256 ictx lists */
   BTRAILER *h_bt[2];                  /**< BTRAILER (current) */
   word8 *h_solve[2], *d_solve[2];     /**< solve seeds */
   word8 *d_map;                       /**< Peach Map */
   int nvml_enabled;                   /**< Flags NVML capable */
} PEACH_CUDA_CTX;


/* pointer to peach CUDA context/s */
static PEACH_CUDA_CTX *PeachCudaCTX;
/* host phash and diff (paged memory txfer) */
static word8 *h_phash, *h_diff;
/* device symbol memory (unique per device) */
__device__ __constant__ static word8 c_phash[SHA256LEN];
__device__ __constant__ static word8 c_diff;

/**
 * @private
 * Perform deterministic (single precision) floating point operations on
 * @a len bytes of @a data (in 4 byte operations).
 * @param data Pointer to data to use in operations
 * @param len Length of @a data to use in operations
 * @param index Peach tile index number
 * @param txf Flag indicates @a data should be transformed by operations
 * @returns 32-bit unsigned operation code for subsequent Peach algo steps
 * @note Operations are guaranteed "deterministic" within the Peach
 * algorithm for all IEEE-754 compliant hardware on "round-to-nearest"
 * rounding mode. This is ensured by the use of CUDA intrinsics:
 * - __fdiv_rn(), __fmul_rn(), __fsub_rn(), __fadd_rn() operations, and
 * - __int2float_rn(), __uint2float_rn() conversions
*/
__device__ static word32 cu_peach_dflops(void *data, size_t len,
   word32 index, int txf)
{
   __constant__ static uint32_t c_float[4] = {
      WORD32_C(0x26C34), WORD32_C(0x14198),
      WORD32_C(0x3D6EC), WORD32_C(0x80000000)
   };
   word8 *bp;
   float *flp, temp, flv;
   int32 operand;
   word32 op;
   unsigned i;
   word8 shift;

   /* process entire length of input data; limit to 4 byte multiples */
   /* len = len - (len & 3); // uncomment if (len % 4 != 0) is expected */
   for (op = i = 0; i < len; i += 4) {
      bp = &((word8 *) data)[i];
      if (txf) {
         /* input data is modified directly */
         flp = (float *) bp;
      } else {
         /* temp variable is modified, input data is unchanged */
         temp = *((float *) bp);
         flp = &temp;
      }
      /* first byte allocated to determine shift amount */
      shift = ((*bp & 7) + 1) << 1;
      /* remaining bytes are selected for 3 different operations based on
       * the first bytes resulting shift on precomputed contants to...
       * ... 1) determine the floating point operation type */
      op += bp[((c_float[0] >> shift) & 3)];
      /* ... 2) determine the value of the operand */
      operand = bp[((c_float[1] >> shift) & 3)];
      /* ... 3) determine the upper most bit of the operand
       *        NOTE: must be performed AFTER the allocation of the operand */
      if (bp[((c_float[2] >> shift) & 3)] & 1) operand ^= c_float[3];
      /* interpret operand as SIGNED integer and cast to float */
      flv = __int2float_rn(operand);
      /* Replace pre-operation NaN with index */
      if (isnan(*flp)) *flp = __uint2float_rn(index);
      /* Perform predetermined floating point operation */
      switch (op & 3) {
         case 3: *flp = __fdiv_rn(*flp, flv);  break;
         case 2: *flp = __fmul_rn(*flp, flv);  break;
         case 1: *flp = __fsub_rn(*flp, flv);  break;
         case 0: *flp = __fadd_rn(*flp, flv);  break;
      }
      /* Replace post-operation NaN with index */
      if (isnan(*flp)) *flp = __uint2float_rn(index);
      /* Add result of the operation to `op` as an array of bytes */
      bp = (word8 *) flp;
      op += bp[0];
      op += bp[1];
      op += bp[2];
      op += bp[3];
   }  /* end for(i = 0; ... */

   return op;
}  /* end cu_peach_dflops() */

/**
 * @private
 * Perform deterministic memory transformations on @a len bytes of @a data.
 * @param data Pointer to data to use in operations
 * @param len Length of @a data to use in operations
 * @param op Operating code from previous Peach algo steps
 * @returns 32-bit unsigned operation code for subsequent Peach algo steps
*/
__device__ static word32 cu_peach_dmemtx(void *data, size_t len, word32 op)
{
   word64 *qp = (word64 *) data;
   word32 *dp = (word32 *) data;
   word8 *bp = (word8 *) data;
   size_t len16, len32, len64, y;
   unsigned i, z;
   word8 temp;

   /* prepare memory pointers and lengths */
   len64 = (len32 = (len16 = len >> 1) >> 1) >> 1;
   /* perform memory transformations multiple times */
   for (i = 0; i < PEACHROUNDS; i++) {
      /* determine operation to use for this iteration */
      op += bp[i];
      /* select "random" transformation based on value of `op` */
      switch (op & 7) {
         case 0:  /* flip the first and last bit in every byte */
            for (z = 0; z < len64; z++) qp[z] ^= WORD64_C(0x8181818181818181);
            for (z <<= 1; z < len32; z++) dp[z] ^= WORD32_C(0x81818181);
            break;
         case 1:  /* Swap bytes */
            for (y = len16, z = 0; z < len16; y++, z++) {
               temp = bp[z]; bp[z] = bp[y]; bp[y] = temp;
            }
            break;
         case 2:  /* 1's complement, all bytes */
            for (z = 0; z < len64; z++) qp[z] = ~qp[z];
            for (z <<= 1; z < len32; z++) dp[z] = ~dp[z];
            break;
         case 3:  /* Alternate +1 and -1 on all bytes */
            for (z = 0; z < len; z++) bp[z] += (z & 1) ? -1 : 1;
            break;
         case 4:  /* Alternate -i and +i on all bytes */
            for (z = 0; z < len; z++) bp[z] += (word8) ((z & 1) ? i : -i);
            break;
         case 5:  /* Replace every occurrence of 104 with 72 */ 
            for (z = 0; z < len; z++) if(bp[z] == 104) bp[z] = 72;
            break;
         case 6:  /* If byte a is > byte b, swap them. */
            for (y = len16, z = 0; z < len16; y++, z++) {
               if(bp[z] > bp[y]) {
                  temp = bp[z]; bp[z] = bp[y]; bp[y] = temp;
               }
            }
            break;
         case 7:  /* XOR all bytes */
            for (y = 0, z = 1; z < len; y++, z++) bp[z] ^= bp[y];
            break;
      } /* end switch(op & 7)... */
   } /* end for(i = 0; ... */

   return op;
}  /* end cu_peach_dmemtx() */

/**
 * @private
 * Perform Nighthash on @a inlen bytes of @a in and place result in @a out.
 * Utilizes deterministic float operations and memory transformations.
 * @param in Pointer to input data
 * @param inlen Length of data from @a in, used in non-transform steps
 * @param index Peach tile index number
 * @param txlen Length of data from @a in, used in transform steps
 * @param out Pointer to location to place resulting hash
*/
__device__ static void cu_peach_nighthash(void *in, size_t inlen,
   word32 index, size_t txlen, void *out)
{
   __constant__ static word64 key32B[4] = { 0, 0, 0, 0 };
   __constant__ static word64 key64B[8] = {
      WORD64_C(0x0101010101010101), WORD64_C(0x0101010101010101),
      WORD64_C(0x0101010101010101), WORD64_C(0x0101010101010101),
      WORD64_C(0x0101010101010101), WORD64_C(0x0101010101010101),
      WORD64_C(0x0101010101010101), WORD64_C(0x0101010101010101),
   };

   /* Perform flops to determine initial algo type.
    * When txlen is non-zero the transformation of input data is enabled,
    * as well as the additional memory transformation process. */
   if (txlen) {
      index = cu_peach_dflops(in, txlen, index, 1);
      index = cu_peach_dmemtx(in, txlen, index);
   } else index = cu_peach_dflops(in, inlen, index, 0);

   /* reduce algorithm selection to 1 of 8 choices */
   switch (index & 7) {
      case 0: cu_blake2b(in, inlen, key32B, 32, out, BLAKE2BLEN256); break;
      case 1: cu_blake2b(in, inlen, key64B, 64, out, BLAKE2BLEN256); break;
      case 2: {
         cu_sha1(in, inlen, out);
         /* SHA1 hash is only 20 bytes long, zero fill remaining... */
         ((word32 *) out)[5] = 0;
         ((word64 *) out)[3] = 0;
         break;
      }
      case 3: cu_sha256(in, inlen, out); break;
      case 4: cu_sha3(in, inlen, out, SHA3LEN256); break;
      case 5: cu_keccak(in, inlen, out, KECCAKLEN256); break;
      case 6: {
         cu_md2(in, inlen, out);
         /* MD2 hash is only 16 bytes long, zero fill remaining... */
         ((word64 *) out)[2] = 0;
         ((word64 *) out)[3] = 0;
         break;
      }
      case 7: {
         cu_md5(in, inlen, out);
         /* MD5 hash is only 16 bytes long, zero fill remaining... */
         ((word64 *) out)[2] = 0;
         ((word64 *) out)[3] = 0;
         break;
      }
   }  /* end switch(algo_type)... */
}  /* end cu_peach_nighthash() */

/**
 * @private
 * Generate a tile of the Peach map.
 * @param index Index number of tile to generate
 * @param tilep Pointer to location to place generated tile
*/
__device__ static void cu_peach_generate(word32 index, word32 *tilep)
{
   int i;

   /* place initial data into seed */
   tilep[0] = index;
   tilep[1] = ((word32 *) c_phash)[0];
   tilep[2] = ((word32 *) c_phash)[1];
   tilep[3] = ((word32 *) c_phash)[2];
   tilep[4] = ((word32 *) c_phash)[3];
   tilep[5] = ((word32 *) c_phash)[4];
   tilep[6] = ((word32 *) c_phash)[5];
   tilep[7] = ((word32 *) c_phash)[6];
   tilep[8] = ((word32 *) c_phash)[7];
   /* perform initial nighthash into first row of tile */
   cu_peach_nighthash(tilep, PEACHGENLEN, index, PEACHGENLEN, tilep);
   /* fill the rest of the tile with the preceding Nighthash result */
   for (i = 0; i < (PEACHTILELEN - 32) / 4; i += 8) {
      tilep[i + 8] = index;
      cu_peach_nighthash(&tilep[i], PEACHGENLEN, index, SHA256LEN,
         &tilep[i + 8]);
   }
}  /* end cu_peach_generate() */

/**
 * @private
 * Perform an index jump using the hash result of the Nighthash function.
 * @param index Index number of (current) tile on Peach map
 * @param nonce Nonce for use as entropy in jump direction
 * @param tilep Pointer to tile data at @a index
 * @returns 32-bit unsigned index of next tile
*/
__device__ void cu_peach_jump(word32 *index, word64 *nonce, word32 *tilep)
{
   word32 seed[PEACHJUMPLEN / 4];
   word32 dhash[SHA256LEN / 4];
   int i;

   /* construct seed for use as Nighthash input for this index on the map */
   ((word64 *) seed)[0] = nonce[0];
   ((word64 *) seed)[1] = nonce[1];
   ((word64 *) seed)[2] = nonce[2];
   ((word64 *) seed)[3] = nonce[3];
   seed[8] = *index;
#pragma unroll
   for (i = 0; i < PEACHTILELEN / 4; i++) {
      seed[i + 9] = tilep[i];
   }

   /* perform nighthash on PEACHJUMPLEN bytes of seed */
   cu_peach_nighthash(seed, PEACHJUMPLEN, *index, 0, dhash);
   /* sum hash as 8x 32-bit unsigned integers */
   *index = (
      dhash[0] + dhash[1] + dhash[2] + dhash[3] +
      dhash[4] + dhash[5] + dhash[6] + dhash[7]
   ) & PEACHCACHELEN_M1;
}  /* end cu_peach_jump() */

/**
 * CUDA kernel for bulk generation of Peach Map tiles.
 * @param d_map Device pointer to location of Peach Map
 * @param offset Index number offset to generate tiles from
 */
__global__ void kcu_peach_build(word8 *d_map, word32 offset)
{
   const word32 index = blockDim.x * blockIdx.x + threadIdx.x + offset;
   if (index < PEACHCACHELEN) {
      cu_peach_generate(index, (word32 *) &d_map[index * PEACHTILELEN]);
   }
}  /* end kcu_peach_build() */

/**
 * CUDA kernel for solving a tokenized haiku as nonce output for Peach proof
 * of work. Combine haiku protocols implemented in the Trigg Algorithm with
 * the memory intensive protocols of the Peach algorithm to generate haiku
 * output as proof of work.
 * @param d_map Device pointer to Peach Map
 * @param d_ictx Device pointer to incomplete hashing contexts
 * @param d_solve Device pointer to location to place nonce on solve
*/
__global__ void kcu_peach_solve(word8 *d_map, SHA256_CTX *d_ictx,
   word8 *d_solve)
{
   word64 nonce[4];
   word8 hash[SHA256LEN];
   SHA256_CTX ictx;
   word32 *x, mario, tid, i;

   tid = (blockIdx.x * blockDim.x) + threadIdx.x;

   /* shift ictx to appropriate location and extract nonce */
#pragma unroll
   for (i = 0; i < sizeof(ictx) / 4; i++) {
      ((word32 *) &ictx)[i] = ((word32 *) &d_ictx[tid])[i];
   }
#pragma unroll
   for (i = 0; i < 8; i++) {
      ((word32 *) nonce)[i] = ((word32 *) &ictx.data[28])[i];
   }
   /* finalise incomplete sha256 hash */
   cu_sha256_final(&ictx, hash);
   /* initialize mario's starting index on the map, bound to PEACHCACHELEN */
   for (mario = hash[0], i = 1; i < SHA256LEN; i++) {
      mario *= hash[i];
   }
   mario &= PEACHCACHELEN_M1;
   /* perform tile jumps to find the final tile x8 */
   for (i = 0; i < PEACHROUNDS; i++) {
      cu_peach_jump(&mario, nonce, (word32 *) &d_map[mario * PEACHTILELEN]);
   }
   /* hash block trailer with final tile */
   cu_sha256_init(&ictx);
   cu_sha256_update(&ictx, hash, SHA256LEN);
   cu_sha256_update(&ictx, &d_map[mario * PEACHTILELEN], PEACHTILELEN);
   cu_sha256_final(&ictx, hash);
   /* Coarse/Fine evaluation checks */
   x = (word32 *) hash;
   for(i = c_diff >> 5; i; i--) if(*(x++) != 0) return;
   if(__clz(__byte_perm(*x, 0, 0x0123)) < (c_diff & 31)) return;

   /* check first to solve with atomic solve handling */
   if(!atomicCAS((int *) d_solve, 0, *((int *) nonce))) {
      ((word64 *) d_solve)[0] = nonce[0];
      ((word64 *) d_solve)[1] = nonce[1];
      ((word64 *) d_solve)[2] = nonce[2];
      ((word64 *) d_solve)[3] = nonce[3];
   }
}  /* end kcu_peach_solve() */

/**
 * CUDA kernel for checking Peach Proof-of-Work. The haiku must be
 * syntactically correct AND have the right vibe. Also, entropy MUST match
 * difficulty.
 * @param ictx Device pointer to incomplete hashing context
 * @param out Pointer to location to place final hash
 * @param eval Evaluation result: VEOK on success, else VERROR
*/
__global__ void kcu_peach_checkhash(SHA256_CTX *ictx, word8 *out,
   word8 *eval)
{
   word64 nonce[4] = { 0 };
   word32 tile[PEACHTILELEN] = { 0 };
   word8 hash[SHA256LEN] = { 0 };
   word32 *x, mario;
   int i;

   /* restricted to a single thread for debug purposes */
   if ((blockDim.x * blockIdx.x) + threadIdx.x > 0) return;

   /* extract nonce */
#pragma unroll
   for (i = 0; i < 8; i++) {
      ((word32 *) nonce)[i] = ((word32 *) &ictx->data[28])[i];
   }
   /* finalise incomplete sha256 hash */
   cu_sha256_final(ictx, hash);
   /* initialize mario's starting index on the map, bound to PEACHCACHELEN */
   for(mario = hash[0], i = 1; i < SHA256LEN; i++) mario *= hash[i];
   mario &= PEACHCACHELEN_M1;
   /* generate and perform tile jumps to find the final tile x8 */
   for (i = 0; i < PEACHROUNDS; i++) {
      cu_peach_generate(mario, tile);
      cu_peach_jump(&mario, nonce, tile);
   }
   /* generate the last tile */
   cu_peach_generate(mario, tile);
   /* hash block trailer with final tile */
   cu_sha256_init(ictx);
   cu_sha256_update(ictx, hash, SHA256LEN);
   cu_sha256_update(ictx, tile, PEACHTILELEN);
   cu_sha256_final(ictx, hash);
   /* pass final hash to out */
   memcpy(out, hash, SHA256LEN);
   /* Coarse/Fine evaluation checks */
   *eval = 1;
   x = (word32 *) hash;
   for(i = c_diff >> 5; i; i--) if(*(x++) != 0) return;
   if(__clz(__byte_perm(*x, 0, 0x0123)) < (c_diff & 31)) return;
   *eval = 0;
}  /* end kcu_peach_checkhash() */

/**
 * Check Peach proof of work algorithm with a CUDA device.
 * @param bt Pointer to block trailer to check
 * @param diff Difficulty to test against entropy of final hash
 * @param out Pointer to location to place final hash, if non-null
 * @returns VEOK on success, else VERROR
*/
int peach_checkhash_cuda(BTRAILER *btp, word8 diff, void *out)
{
   SHA256_CTX *d_ictx, ictx;
   word8 *d_hash, *d_eval;
   word8 eval = 0;
   int count;

   cuCHK(hipGetDeviceCount(&count), NULL, return (-1));
   if (count < 1) {
      pfatal("No CUDA devices...");
      return -1;
   }
   cuCHK(hipSetDevice(0), NULL, return (-1));
   cuCHK(hipMalloc(&d_ictx, sizeof(SHA256_CTX)), NULL, return (-1));
   cuCHK(hipMalloc(&d_hash, SHA256LEN), NULL, return (-1));
   cuCHK(hipMalloc(&d_eval, 1), NULL, return (-1));
   cuCHK(hipMemset(d_eval, 0xff, 1), NULL, return (-1));
   /* prepare intermediate state for next round */
   sha256_init(&ictx);
   sha256_update(&ictx, btp, 124);
   /* transfer phash to device */
   cuCHK(hipMemcpyToSymbol(HIP_SYMBOL(c_diff), btp->difficulty, 1, 0,
      hipMemcpyHostToDevice), NULL, return (-1));
   /* transfer phash to device */
   cuCHK(hipMemcpyToSymbol(HIP_SYMBOL(c_phash), btp->phash, SHA256LEN, 0,
      hipMemcpyHostToDevice), NULL, return (-1));
   /* transfer ictx to device */
   cuCHK(hipMemcpy(d_ictx, &ictx, sizeof(SHA256_CTX),
      hipMemcpyHostToDevice), NULL, return (-1));
   /* launch kernel to check Peach */
   kcu_peach_checkhash<<<1, 1>>>(d_ictx, d_hash, d_eval);
   cuCHK(hipGetLastError(), NULL, return (-1));
   /* retrieve hash/eval data */
   cuCHK(hipMemcpy(out, d_hash, SHA256LEN,
      hipMemcpyDeviceToHost), NULL, return (-1));
   cuCHK(hipMemcpy(&eval, d_eval, 1,
      hipMemcpyDeviceToHost), NULL, return (-1));
   /* wait for device to finish */
   cuCHK(hipDeviceSynchronize(), NULL, return (-1));
   /* free memory */
   cuCHK(hipFree(d_ictx), NULL, return (-1));
   cuCHK(hipFree(d_hash), NULL, return (-1));
   cuCHK(hipFree(d_eval), NULL, return (-1));
   /* return */
   return (int) eval;
}  /* end peach_checkhash_cuda() */

/**
 * (re)Initialize a device context with a CUDA device.
 * @param devp Pointer to DEVICE_CTX to initialize
 * @param id Index of CUDA device to initialize to DEVICE_CTX
 * @returns VEOK on success, else VERROR
 * @note The `id` parameter of the DEVICE_CTX must be set to an appropriate
 * CUDA device number. If not performing a re-initialization, recommend
 * using peach_init_cuda() first.
*/
int peach_init_cuda_device(DEVICE_CTX *devp, int id)
{
   static int nvml_initialized = 0;
   static unsigned nvml_count = 0;

   struct hipDeviceProp_t props;
   nvmlPciInfo_t pci;
   nvmlDevice_t *nvmlp;
   size_t ictxlen;
   unsigned i, gen, width;

   if (nvml_initialized == 0) {
      /* set nvml initialized */
      nvml_initialized = 1;
      /* initialize nvml */
      if (nvmlInit() != NVML_SUCCESS) {
         nvml_count = pdebug("Unable to initialize NVML");
         plog("No NVML devices detected...");
      } else if (nvmlDeviceGetCount(&nvml_count) != NVML_SUCCESS) {
         nvml_count = pdebug("Unable to obtain NVML count");
         plog("No NVML devices detected...");
      }
   }

   devp->id = id;
   devp->type = CUDA_DEVICE;
   nvmlp = &(PeachCudaCTX[id].nvml_device);
   /* get CUDA properties for verification with nvml device */
   if (hipGetDeviceProperties(&props, id) != hipSuccess) {
      perr("hipGetDeviceProperties(%d)", id);
   } else {
      /* scan nvml devices for match */
      PeachCudaCTX[id].nvml_enabled = 0;
      for (i = 0; i < nvml_count; i++) {
         memset(nvmlp, 0, sizeof(nvmlDevice_t));
         if (nvmlDeviceGetHandleByIndex(i, nvmlp) == NVML_SUCCESS ||
            (nvmlDeviceGetPciInfo(*nvmlp, &pci) == NVML_SUCCESS) ||
            (pci.device == props.pciDeviceID) ||
            (pci.domain == props.pciDomainID) ||
            (pci.bus == props.pciBusID)) {
            /* obtain link gen/width */
            if (nvmlDeviceGetCurrPcieLinkGeneration(*nvmlp, &gen)
                  != NVML_SUCCESS) gen = 0;
            if (nvmlDeviceGetCurrPcieLinkWidth(*nvmlp, &width)
                  != NVML_SUCCESS) width = 0;
            PeachCudaCTX[id].nvml_enabled = 1;
            break;
         }
      }
      /* store GPU name, PCI Id and gen info in nameId */
      snprintf(devp->nameId, sizeof(devp->nameId),
         "%04u:%02u:%02u:%.128s Gen%1ux%02u", props.pciDomainID,
         props.pciDeviceID, props.pciBusID, props.name, gen, width);
   }
   /* set context to CUDA id */
   cuCHK(hipSetDevice(id), devp, return VERROR);
   /* set CUDA configuration for device */
   if (hipOccupancyMaxPotentialBlockSize(&(devp->grid), &(devp->block),
         kcu_peach_solve, 0, 0) != hipSuccess) {
      pdebug("cudaOccupancy~BlockSize(%d) failed...", id);
      pdebug("Using conservative defaults for <<<512/128>>>");
      devp->grid = 512;
      devp->block = 128;
   }
   /* calculate total threads and ictxlist size */
   devp->threads = devp->grid * devp->block;
   ictxlen = sizeof(SHA256_CTX) * devp->threads;
   /* create streams for device */
   cuCHK(hipStreamCreate(&(PeachCudaCTX[id].stream[0])), devp, return VERROR);
   cuCHK(hipStreamCreate(&(PeachCudaCTX[id].stream[1])), devp, return VERROR);
   /* allocate pinned host memory for host/device transfers */
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_solve[0]), 32), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_solve[1]), 32), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_ictx[0]), ictxlen), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_ictx[1]), ictxlen), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_bt[0]), sizeof(BTRAILER)), devp, return VERROR);
   cuCHK(hipHostMalloc(&(PeachCudaCTX[id].h_bt[1]), sizeof(BTRAILER)), devp, return VERROR);
   /* allocate device memory for host/device transfers */
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_solve[0]), 32), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_solve[1]), 32), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_ictx[0]), ictxlen), devp, return VERROR);
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_ictx[1]), ictxlen), devp, return VERROR);
   /* allocate memory for Peach map on device */
   cuCHK(hipMalloc(&(PeachCudaCTX[id].d_map), PEACHMAPLEN), devp, return VERROR);
   /* clear device/host allocated memory */
   hipMemsetAsync(PeachCudaCTX[id].d_ictx[0], 0, ictxlen, hipStreamDefault);
   cuCHK(hipGetLastError(), devp, return VERROR);
   hipMemsetAsync(PeachCudaCTX[id].d_ictx[1], 0, ictxlen, hipStreamDefault);
   cuCHK(hipGetLastError(), devp, return VERROR);
   hipMemsetAsync(PeachCudaCTX[id].d_solve[0], 0, 32, hipStreamDefault);
   cuCHK(hipGetLastError(), devp, return VERROR);
   hipMemsetAsync(PeachCudaCTX[id].d_solve[1], 0, 32, hipStreamDefault);
   cuCHK(hipGetLastError(), devp, return VERROR);
   memset(PeachCudaCTX[id].h_bt[0], 0, sizeof(BTRAILER));
   memset(PeachCudaCTX[id].h_bt[1], 0, sizeof(BTRAILER));
   memset(PeachCudaCTX[id].h_ictx[0], 0, ictxlen);
   memset(PeachCudaCTX[id].h_ictx[1], 0, ictxlen);
   memset(PeachCudaCTX[id].h_solve[0], 0, 32);
   memset(PeachCudaCTX[id].h_solve[1], 0, 32);

   return VEOK;
}  /* end peach_init_cuda_device() */

/**
 * Initialize a DEVICE_CTX list with CUDA devices for solving the Peach
 * proof of work algorithm.
 * @param devlist Pointer to DEVICE_CTX list to initialize
 * @param max Maximum number of CUDA devices to initialize
 * @returns number of CUDA devices available for initialization
 * @note It is possible to have "some" CUDA devices fail to initialize.
*/
int peach_init_cuda(DEVICE_CTX devlist[], int max)
{
   static int cuda_initialized = 0;
   static int cuda_num = 0;

   int id;

   /* avoid re-initialization attempts */
   if (cuda_initialized) return cuda_num;

   /* check for cuda driver and devices */
   switch(hipGetDeviceCount(&cuda_num)) {
      case hipErrorNoDevice:
         return plog("No CUDA devices detected...");
      case hipErrorInsufficientDriver:
         pfatal("Insufficient CUDA Driver. Update display drivers...");
         return 0;
      case hipSuccess:
         if (cuda_num > max) {
            cuda_num = max;
            plog("CUDA Devices: %d (limited)\n", cuda_num);
            perr("CUDA device count EXCEEDED maximum count parameter!");
            pwarn("Some CUDA devices will not be utilized.");
            plog("Please advise developers if this is an issue...");
         }
         break;
      default:
         pfatal("Unknown CUDA initialization error occured...");
         return 0;
   }

   /* set initialized */
   cuda_initialized = 1;
   if (cuda_num < 1) return (cuda_num = 0);

   /* allocate memory for PeachCudaCTX */
   PeachCudaCTX =
      (PEACH_CUDA_CTX *) malloc(sizeof(PEACH_CUDA_CTX) * cuda_num);

   /* allocate pinned host memory for data consistant across devices */
   cuCHK(hipHostMalloc(&h_phash, SHA256LEN), NULL, return (cuda_num = 0));
   cuCHK(hipHostMalloc(&h_diff, 1), NULL, return (cuda_num = 0));
   memset(h_phash, 0, SHA256LEN);
   memset(h_diff, 0, 1);

   /* initialize device contexts for CUDA num devices */
   for(id = 0; id < cuda_num; id++) {
      peach_init_cuda_device(&devlist[id], id);
   }

   return cuda_num;
}  /* end peach_init_cuda() */

/**
 * Try solve for a tokenized haiku as nonce output for Peach proof of work
 * on CUDA devices. Combine haiku protocols implemented in the Trigg
 * Algorithm with the memory intensive protocols of the Peach algorithm to
 * generate haiku output as proof of work.
 * @param dev Pointer to DEVICE_CTX to perform work with
 * @param bt Pointer to block trailer to solve for
 * @param diff Difficulty to test against entropy of final hash
 * @param btout Pointer to location to place solved block trailer
 * @returns VEOK on solve, else VERROR
*/
int peach_solve_cuda(DEVICE_CTX *dev, BTRAILER *bt, word8 diff, BTRAILER *btout)
{
   int i, id, sid, grid, block;
   PEACH_CUDA_CTX *P;
   nvmlReturn_t nr;
   size_t ictxlen;

   /* check for GPU failure */
   if (dev->status == DEV_FAIL) return VERROR;

   id = dev->id;
   P = &PeachCudaCTX[id];
   /* set/check cuda device */
   cuCHK(hipSetDevice(id), dev, return VERROR);
   cuCHK(hipGetLastError(), dev, return VERROR);

   /* ensure initialization is complete */
   if (dev->status == DEV_NULL) {
      if (hipStreamQuery(hipStreamDefault) != hipSuccess) return VERROR;
      /* set next action to build Peach map */
      dev->status = DEV_INIT;
      dev->last_work = time(NULL);
      dev->total_work = 0;
      dev->work = 0;
   }

   /* build peach map */
   if (dev->status == DEV_INIT) {
      /* ensure both streams have finished */
      if (hipStreamQuery(P->stream[1]) == hipSuccess
         && hipStreamQuery(P->stream[0]) == hipSuccess) {
         /* build peach map -- init */
         if (dev->work == 0) {
            /* clear any late solves */
            memset(P->h_solve[0], 0, 32);
            memset(P->h_solve[1], 0, 32);
            hipMemcpyAsync(P->d_solve[0], P->h_solve[0], 32,
               hipMemcpyHostToDevice, P->stream[0]);
            cuCHK(hipGetLastError(), dev, return VERROR);
            hipMemcpyAsync(P->d_solve[1], P->h_solve[1], 32,
               hipMemcpyHostToDevice, P->stream[0]);
            cuCHK(hipGetLastError(), dev, return VERROR);
            /* update block trailer */
            memcpy(P->h_bt[0], bt, sizeof(BTRAILER));
            memcpy(P->h_bt[1], bt, sizeof(BTRAILER));
            /* ensure phash is set */
            memcpy(h_phash, bt->phash, SHA256LEN);
            /* asynchronous copy to phash and difficulty symbols */
            hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), h_phash, SHA256LEN, 0,
               hipMemcpyHostToDevice, P->stream[0]);
            cuCHK(hipGetLastError(), dev, return VERROR);
            /* update h_diff from function diff parameter */
            if (*h_diff != diff) *h_diff = diff;
            /* if h_diff is zero (0) or greater than bt, use bt */
            if (*h_diff == 0 || *h_diff > bt->difficulty[0]) {
               hipMemcpyToSymbolAsync(HIP_SYMBOL(c_diff), P->h_bt[0]->difficulty,
                  1, 0, hipMemcpyHostToDevice, P->stream[0]);
               cuCHK(hipGetLastError(), dev, return VERROR);
            } else {
               hipMemcpyToSymbolAsync(HIP_SYMBOL(c_diff), h_diff,
                  1, 0, hipMemcpyHostToDevice, P->stream[0]);
               cuCHK(hipGetLastError(), dev, return VERROR);
            }
         }
         /* build peach map -- build */
         if (dev->work < PEACHCACHELEN) {
            /* set CUDA configuration for generating peach map */
            if (hipOccupancyMaxPotentialBlockSize(&grid, &block,
                  kcu_peach_build, 0, 0) != hipSuccess) {
               pdebug("cudaOccupancy~BlockSize(%d) failed...", id);
               pdebug("Using conservative defaults, <<<128/128>>>");
               grid = 128;
               block = 128;
            }
            /* launch kernel to generate map */
            kcu_peach_build<<<grid, block, 0, P->stream[0]>>>
               (P->d_map, (word32) dev->work);
            cuCHK(hipGetLastError(), dev, return VERROR);
            /* update build progress */
            dev->work += grid * block;
         } else {
            /* build is complete */
            dev->last_work = time(NULL);
            dev->status = DEV_IDLE;
            dev->work = 0;
         }
      }
   }

   /* check for solvable work in block trailer */
   if (dev->status == DEV_IDLE && get32(bt->tcount)) {
      /* remain idle if btout contains a solve for current block number */
      if (cmp64(bt->bnum, btout->bnum)) dev->status = DEV_WORK;
   }

   /* solve work in block trailer */
   if (dev->status == DEV_WORK) {
      for(sid = 0; sid < 2; sid++) {
         if (hipStreamQuery(P->stream[sid]) != hipSuccess) continue;
         /* check trailer for block update */
         if (memcmp(h_phash, bt->phash, HASHLEN)) {
            dev->status = DEV_INIT;
            dev->work = 0;
            break;
         }
         /* switch to idle mode if no transactions or already solved bnum */
         if (get32(bt->tcount) == 0 || cmp64(bt->bnum, btout->bnum) == 0) {
            dev->status = DEV_IDLE;
            dev->work = 0;
            break;
         }
         /* check for solves */
         if (*(P->h_solve[sid])) {
            /* move solved nonce */
            memcpy(P->h_bt[sid]->nonce, P->h_solve[sid], 32);
            /* clear solve from host/device */
            memset(P->h_solve[sid], 0, 32);
            hipMemcpyAsync(P->d_solve[sid], P->h_solve[sid], 32,
               hipMemcpyHostToDevice, P->stream[sid]);
            cuCHK(hipGetLastError(), dev, return VERROR);
            memcpy(btout, P->h_bt[sid], sizeof(BTRAILER));
            /* return a solve */
            return VEOK;
         }
         /* ensure block trailer is updated */
         memcpy(P->h_bt[sid], bt, BTSIZE);
         /* generate nonce directly into block trailer */
         trigg_generate_fast(P->h_bt[sid]->nonce);
         trigg_generate_fast(P->h_bt[sid]->nonce + 16);
         /* prepare intermediate state for next round */
         sha256_init(P->h_ictx[sid]);
         sha256_update(P->h_ictx[sid], P->h_bt[sid], 124);
         /* duplicate intermediate state with random second seed */
         for(i = 1; i < dev->threads; i++) {
            memcpy(&(P->h_ictx[sid][i]), P->h_ictx[sid], sizeof(SHA256_CTX));
            trigg_generate_fast(P->h_ictx[sid][i].data + 44);
         }
         /* transfer ictx to device */
         ictxlen = sizeof(SHA256_CTX) * dev->threads;
         hipMemcpyAsync(P->d_ictx[sid], P->h_ictx[sid], ictxlen,
            hipMemcpyHostToDevice, P->stream[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* launch kernel to solve Peach */
         kcu_peach_solve<<<dev->grid, dev->block, 0, P->stream[sid]>>>
            (P->d_map, P->d_ictx[sid], P->d_solve[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* retrieve solve seed */
         hipMemcpyAsync(P->h_solve[sid], P->d_solve[sid], 32,
            hipMemcpyDeviceToHost, P->stream[sid]);
         cuCHK(hipGetLastError(), dev, return VERROR);
         /* increment progress counters */
         dev->total_work += dev->threads;
         dev->work += dev->threads;
      }
   }

   /* power and temperature monitoring (1 second interval) */
   if (P->nvml_enabled && difftime(time(NULL), dev->last_monitor)) {
      dev->last_monitor = time(NULL);
      /* get GPU device power */
      unsigned int fan;
      nr = nvmlDeviceGetFanSpeed(P->nvml_device, &fan);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) fan speed: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->fan = fan;
      /* get GPU device power */
      unsigned int power;
      nr = nvmlDeviceGetPowerUsage(P->nvml_device, &power);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) power usage: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->pow = power / 1000;
      /* get GPU device temperature */
      unsigned int temperature;
      nr = nvmlDeviceGetTemperature(P->nvml_device, NVML_TEMPERATURE_GPU,
         &temperature);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) temperature: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->temp = temperature;
      /* get GPU device utilization */
      nvmlUtilization_t utilization;
      nr = nvmlDeviceGetUtilizationRates(P->nvml_device, &utilization);
      if (nr != NVML_SUCCESS) {
         perr("nvml(%d) utilization rates: %s\n", id, nvmlErrorString(nr));
         memset(&(P->nvml_device), 0, sizeof(nvmlDevice_t));
         P->nvml_enabled = 0;
      } else dev->util = utilization.gpu;
   }

   return VERROR;
}  /* end peach_solve_cuda() */

/* end include guard */
#endif
